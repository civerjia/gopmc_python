#include "hip/hip_runtime.h"
//#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>

#include <cmath>
// #include <array>
#include <vector>
//#include <cmath>// cannot use in cu files
#include <iostream>
#include <stdio.h>
#include "mex.h"
// #include <omp.h>
#include <string>
#include <fstream>      // std::ofstream
#include <filesystem>

using T = float;
using host_vec = thrust::host_vector<float>;
using device_vec = thrust::device_vector<float>;

__constant__ const T const_1_PI = 0.318309886183791;
__constant__ const T const_1_SQRT_2PI = 0.398942280401433;
// 1/sqrt(2*pi), 1D gauss constant
__constant__ const T const_1_2PI = 0.159154943091895;
// 1/(2*pi), 2D gauss constant
__constant__ int constmem_Nsize[5];
// declare constant memory, Nx,Ny,Nz,N_gaussian,N_para

void set_constant_mem(int Nx, int Ny, int Nz, int N_gaussian, int N_para)
{
    int cNsize[5] = { Nx,Ny,Nz,N_gaussian,N_para }; // copy host data to constant memory
    //hipError_t mem_err;
    //mem_err = hipMemcpyToSymbol(HIP_SYMBOL(constmem_Nsize), &cNsize, sizeof(int) * 5);
    hipMemcpyToSymbol(HIP_SYMBOL(constmem_Nsize), &cNsize, sizeof(int) * 5);
}

__inline__ __device__ T gauss1d(T x, T A, T mu, T sigma)
{
    T c{ const_1_SQRT_2PI / sigma };
    T xnew = (x - mu) / sigma;
    return A * c * expf(-0.5 * xnew * xnew);
}

__inline__ __device__ T gauss2d(T x, T y, T A, T mux, T muy, T sigma)
{// isotropic 2d gaussian function
    if ((sigma < 1e-6))
    {
        return 0.0;
    }
    else
    {
        T half_1_sigma2 = 1.0 / (2.0 * sigma * sigma);
        T c = const_1_PI * half_1_sigma2;
        T xnew = x - mux;
        T ynew = y - muy;
        return A * c * expf(-half_1_sigma2 * (xnew * xnew + ynew * ynew));
    }
}
__inline__ __device__ T mvn2d(T x, T y, T A, T mux, T muy, T sigma1, T sigma2, T beta)
{
    if ((sigma1 < 1e-6) | (sigma2 < 1e-6))
    {
        return 0.0f;
    }
    else
    {
        T sigma11 = sigma1 * sigma1;
        T sigma12 = 0.0f;
        T sigma21 = 0.0f;
        T sigma22 = sigma2 * sigma2;
        T det = sigma11 * sigma22 - sigma12 * sigma21;

        // invsere of corvariance matrix 
        T a = sigma22 / det;
        T b = -sigma12 / det;
        T c = -sigma21 / det;
        T d = sigma11 / det;

        // v = Rot([x - mux;y - muy])
        T v1 = cosf(beta) * (x - mux) - sinf(beta) * (y - muy);
        T v2 = sinf(beta) * (x - mux) + cosf(beta) * (y - muy);
        // v' * M^-1 * v
        T u1 = a * v1 + b * v2;
        T u2 = c * v1 + d * v2;
        T exponant = -0.5 * (v1 * u1 + v2 * u2);
        T scale = (A * const_1_2PI) / sqrt(abs(det));

        return scale * expf(exponant);
    }
}
__global__ void dose3d_N_iso(T* X, T* Y, T* para, T* dose3d)
{
    int nx = blockIdx.x * blockDim.x + threadIdx.x;
    int ny = blockIdx.y * blockDim.y + threadIdx.y;
    int nz = blockIdx.z * blockDim.z + threadIdx.z;
    int Nx = constmem_Nsize[0];
    int Ny = constmem_Nsize[1];
    int Nz = constmem_Nsize[2];
    int N_gaussian = constmem_Nsize[3];
    if (nz < Nz & ny < Ny & nx < Nx)
    {
        int idx3d{ nx + ny * Nx + nz * (Nx * Ny) };
        T x{ X[nx] };
        T y{ Y[ny] };
        T temp{};
        for (int ng = 0; ng < N_gaussian; ++ng)
        {
            T A1 = para[nz * N_gaussian * 4 + 4 * ng];
            T mux1 = para[nz * N_gaussian * 4 + 4 * ng + 1];
            T muy1 = para[nz * N_gaussian * 4 + 4 * ng + 2];
            T sigma1 = para[nz * N_gaussian * 4 + 4 * ng + 3];
            temp += gauss2d(x, y, A1, mux1, muy1, sigma1);
        }
        dose3d[idx3d] = temp;
    }
}


__global__ void dose3d_N(T* X, T* Y, T* para, T* dose3d)
{
    int nx = blockIdx.x * blockDim.x + threadIdx.x;
    int ny = blockIdx.y * blockDim.y + threadIdx.y;
    int nz = blockIdx.z * blockDim.z + threadIdx.z;
    int Nx = constmem_Nsize[0];
    int Ny = constmem_Nsize[1];
    int Nz = constmem_Nsize[2];
    int N_gaussian = constmem_Nsize[3];
    // __shared__ X_shared[512];
    // __shared__ Y_shared[512];
    if (nx >= Nx || ny >= Ny || nz >= Nz) return;

    // X_shared[nx] = X[nx];
    // Y_shared[ny] = Y[ny];
    // __syncthreads();
    // 
    int idx3d{ nx + ny * Nx + nz * (Nx * Ny) };
    T x{ X[nx] };
    T y{ Y[ny] };
    T temp{};
    for (int ng = 0; ng < N_gaussian; ++ng)
    {
        T A = para[nz * N_gaussian * 6 + 6 * ng];
        T mux = para[nz * N_gaussian * 6 + 6 * ng + 1];
        T muy = para[nz * N_gaussian * 6 + 6 * ng + 2];
        T sigma1 = para[nz * N_gaussian * 6 + 6 * ng + 3];
        T sigma2 = para[nz * N_gaussian * 6 + 6 * ng + 4];
        T beta = para[nz * N_gaussian * 6 + 6 * ng + 5];
        temp += mvn2d(x, y, A, mux, muy, sigma1, sigma2, beta);
    }
    dose3d[idx3d] = temp;

}

void cpu_interface(host_vec X, host_vec Y, host_vec para, host_vec& dose3D, int Nx, int Ny, int Nz, int N_para, int N_gaussian)
{
    // copy data to device
    device_vec X_dev = X;
    device_vec Y_dev = Y;
    device_vec para_dev = para;
    device_vec dose3D_dev = dose3D;
    // cast to raw pointer
    T* X_dev_ptr = thrust::raw_pointer_cast(X_dev.data());
    T* Y_dev_ptr = thrust::raw_pointer_cast(Y_dev.data());
    T* para_dev_ptr = thrust::raw_pointer_cast(para_dev.data());
    T* dose3D_dev_ptr = thrust::raw_pointer_cast(dose3D_dev.data());

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((Nx - 1 + threadsPerBlock.x) / threadsPerBlock.x, (Ny - 1 + threadsPerBlock.y) / threadsPerBlock.y,
        (Nz - 1 + threadsPerBlock.z) / threadsPerBlock.z);
    if (N_gaussian * 6 == N_para)
    {
        dose3d_N << <numBlocks, threadsPerBlock >> > (X_dev_ptr, Y_dev_ptr, para_dev_ptr, dose3D_dev_ptr);
    }
    else if (N_gaussian * 4 == N_para)
    {
        dose3d_N_iso << <numBlocks, threadsPerBlock >> > (X_dev_ptr, Y_dev_ptr, para_dev_ptr, dose3D_dev_ptr);
    }
    else
    {
        std::cout << "wrong N_para not equal 6 * N_gaussian or 4 * N_gaussian \n";
    }
    // copy data back to host
    thrust::copy(dose3D_dev.begin(), dose3D_dev.end(), dose3D.begin());
}
// save data to binary file
void save_dat(std::string filename, host_vec vec)
{
    std::ofstream outdata; // outdata is like cin
    std::string fmt{ ".dat" };
    outdata.open(filename + fmt, std::ofstream::binary | std::ofstream::out); // opens the file
    outdata.write((char*)vec.data(), vec.size() * sizeof(T));
    outdata.close();
}
void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {
    T *X;
    T *Y;
    T *para;
    X = (T*)mxGetPr(prhs[0]);
    Y = (T*)mxGetPr(prhs[1]);
    para = (T*)mxGetPr(prhs[2]);

    const mwSize *dim_X = mxGetDimensions(prhs[0]);
    const mwSize *dim_Y = mxGetDimensions(prhs[1]);
    const mwSize *dim_para = mxGetDimensions(prhs[2]);
    int Nx = static_cast<int>(dim_X[0]*dim_X[1]);
    int Ny = static_cast<int>(dim_Y[0]*dim_Y[1]);
    int N_para = static_cast<int>(dim_para[0]*dim_para[1]);

    int Nz = static_cast<int>(*mxGetPr(prhs[3]));
    int N_gaussian = static_cast<int>(*mxGetPr(prhs[4]));

    set_constant_mem(Nx, Ny, Nz, N_gaussian, N_para);

    host_vec X_vec(Nx), Y_vec(Ny);
    thrust::copy(X, X + Nx, X_vec.begin());
    thrust::copy(Y, Y + Ny, Y_vec.begin());

    host_vec para_vec(N_para);
    thrust::copy(para, para + N_para, para_vec.begin());

    const mwSize size[3]{ mwSize(Nx), mwSize(Ny), mwSize(Nz) };
    int64_t size3d = int64_t(Nx)* int64_t(Ny)* int64_t(Nz);
    plhs[0] = mxCreateNumericArray(3, size, mxSINGLE_CLASS, mxREAL);
    T* dose3d_ptr{};
    dose3d_ptr = (T*)mxGetPr(plhs[0]);
    host_vec dose3d(size3d);

    
    cpu_interface(X_vec, Y_vec, para_vec, dose3d, Nx, Ny, Nz, N_para, N_gaussian);
    thrust::copy(dose3d.begin(), dose3d.end(), dose3d_ptr);

//     std::string filename = "dose3D";
//     save_dat(filename,dose3d);
}
